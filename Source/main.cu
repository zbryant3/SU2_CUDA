#include "hip/hip_runtime.h"
//*****************************************************
// Usage: Performs SU[2] simulations utilizing        *
//  monte carlo calculations performed on the GPU.    *
//                                                    *
// Author: Zachariah Bryant                           *
//*****************************************************


//**************
//   Headers   *
//**************
#include <iostream>
#include <fstream>
#include <string.h>
#include <vector>

//Contains class wrap for SU model to be performed on the gpu
#include "./Headers/LattiCuda.cuh"


//**************************************
//   Definition of all the variables   *
//**************************************
#define LATTSIZE 8
#define BETA 2.0




using namespace std;


//**********************
//    Main Function    *
//**********************
int main()
{
        LattiCuda model(LATTSIZE, BETA);

        fstream File;
        double temp;

        File.open("AvgPlaq_vs_Equilibration.dat", ios::out | ios::trunc);

        for(int i = 0; i < 10; i++){
          temp = model.AvgPlaquette();
          cout << temp << "\n";
          File << i << " " << temp << "\n";
          File.flush();
          model.Equilibrate();
        }
        File.close();


        return 0;
}
