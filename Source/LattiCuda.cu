#include "hip/hip_runtime.h"
#include "./Headers/LattiCuda.cuh"
#include "./Headers/LattiCuda_Device.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>


using namespace std;

//*************************************
//    Global Variable Declarations    *
//*************************************
 
/**
 * Constant Variables for device code
 * @var   d_size    - Size of the lattice
 * @var   d_beta    - Value of Beta
 */
__constant__ int d_size;
__constant__ double d_beta;


//*********************************
//      GPU Kernal Functions      *
//*********************************

/**
 * Initializes all the links on the lattice to the unit matrix
 */
__global__ void
GPU_Initialize(thrust::complex<double> *d_lattice){

        LattiCuda_Device device(&d_size, &d_beta, d_lattice);

        device.Initialize();
};


/**
 * Equilibrates the lattice using the GPU.
 * @param  d_lattice - Pointer to the lattice in device memory
 */
__global__ void
GPU_Equilibrate(thrust::complex<double> *d_lattice, int tdim){

        //Shared sublattice memory with size determined at kernal launch
        extern __shared__ thrust::complex<double> sub_lattice[];

        LattiCuda_Device device(&d_size, &d_beta, d_lattice, sub_lattice, tdim);

        device.Equilibrate();

};



/**
 * Gets the average plaquette of the lattice
 */
__global__ void
GPU_AvgPlaquette(thrust::complex<double> *d_lattice, int tdim, double *d_plaq, double *d_iter){

        //Shared sublattice memory with size determined at kernal launch
        //extern __shared__ thrust::complex<double> sub_lattice[];

        LattiCuda_Device device(&d_size, &d_beta, d_lattice, NULL, tdim);

        device.AvgPlaquette(d_plaq, d_iter);

};


//*******************************
//    Private Member Functions  *
//*******************************

/**
 * Initializes all the links on the lattice to the unit matrix
 */
__host__ void
LattiCuda::Initialize(){

        //                X           Y split            Z split
        dim3 in_Threads(h_size, h_size/(h_size/2), h_size/(h_size/2));

        //  sizeofsplit:   Y           Z         T-Dimension
        dim3 in_Blocks((h_size/2), (h_size/2), h_size);

        GPU_Initialize<<<in_Blocks,in_Threads>>>(d_lattice);
};



//*******************************
//    Public Member Functions   *
//*******************************

/**
 * Constructor for the Lattice QCD wrapper
 * @param   LattSize  - Size of desired lattice
 * @param   inBeta    - Beta value
 */
__host__
LattiCuda::LattiCuda(int LattSize, double inBeta){

        //Construct Host Variables
        h_size = LattSize;
        h_beta = inBeta;
        memsize = h_size*h_size*h_size*h_size*4*4;

        //Create Host Lattice
        h_lattice = new thrust::complex<double>[memsize];

        //Create Device Lattice
        hipMalloc((void**)&d_lattice, memsize*sizeof(thrust::complex<double>));

        //Construct Constant Device Variables
        hipMemcpyToSymbol(HIP_SYMBOL(d_size), &h_size, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &h_beta, sizeof(double));

        //Initialize the lattice on creation
        Initialize();

};


/**
 * Destructor for the Lattice QCD wrapper
 */
__host__
LattiCuda::~LattiCuda(){

        delete[] h_lattice;

        hipFree(d_lattice);

};


/**
 * Equilibrates the lattice
 */
__host__ void
LattiCuda::Equilibrate(){

        int half = h_size/2;

        //Dimensions for the kernal
        dim3 Threads(h_size/half, h_size/half, h_size/half);
        dim3 Blocks(half, half, half);

        //Max shared size is 49152
        int sharedsize = ((h_size)/(half) + 2) * ((h_size)/(half) + 2)
                         * ((h_size)/(half) + 2) * 768;

        //Ensures shared size isnt too large
        if(sharedsize > 49152) {
                cout << "Shared memory size too large. Exiting... \n \n";
                exit(EXIT_FAILURE);
        }

        //Checkerboard pattern for T dimension
        for(int offset = 0; offset <= 1; offset++) {
                for(int tdim = 0; tdim < half; tdim++) {
                        GPU_Equilibrate<<<Blocks, Threads, sharedsize>>>(d_lattice, (tdim)*2 + offset);
                }
                hipDeviceSynchronize();
        }

};



/**
 * Gets the value of the average plaquette of the lattice
 * @return double - Average Plaquette
 */
__host__ double
LattiCuda::AvgPlaquette(){

        int half = h_size/2;


        //Array to hold total avg plaquett per thread and total amount of iterations
        double h_plaq[h_size*h_size*h_size*h_size];
        double h_iter[h_size*h_size*h_size*h_size];
        double *d_plaq;
        double *d_iter;

        hipMalloc((void**)&d_plaq, sizeof(double)*h_size*h_size*h_size*h_size);
        hipMalloc((void**)&d_iter, sizeof(double)*h_size*h_size*h_size*h_size);

        //Dimensions for the kernal
        dim3 Threads(h_size/half, h_size/half, h_size/half);
        dim3 Blocks(half, half, half);

        /*
        //Max shared size is 49152
        int sharedsize = ((h_size)/(half) + 2) * ((h_size)/(half) + 2)
                         * ((h_size)/(half) + 2) * 768;

        //Ensures shared size isnt too large
        if(sharedsize > 49152) {
                cout << "Shared memory size too large. Exiting... \n \n";
                exit(EXIT_FAILURE);
        }
        */

        for(int tdim = 0; tdim < h_size; tdim++) {
                GPU_AvgPlaquette<<<Blocks, Threads>>>
                (d_lattice, tdim, d_plaq, d_iter);
                hipDeviceSynchronize();
        }

        hipMemcpy(h_plaq, d_plaq, sizeof(double)*h_size*h_size*h_size*h_size, hipMemcpyDeviceToHost);
        hipMemcpy(h_iter, d_iter, sizeof(double)*h_size*h_size*h_size*h_size, hipMemcpyDeviceToHost);


        double totplaq{0};
        double totiter{0};
        for(int i = 0; i < h_size*h_size*h_size*h_size; i++){
          totplaq += h_plaq[i];
          totiter += h_iter[i];
          //cout << h_plaq[i] << "\n";
        }

        hipFree(d_plaq);
        hipFree(d_iter);

        return (totplaq/totiter);
};
